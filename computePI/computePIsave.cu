
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <time.h>
//#include <cutil_inline.h>
using namespace std;

//*****************************************//
//以下两部分将在设备上编译 由__global__标识；
template<typename T> __global__ void reducePI1(T* __restrict__ d_sum, int num){
//__restrict__ 是说从只读缓存中读取该数据，会有什么优势呢？
//printf("blockIdx.x is %d\n",blockIdx.x);//线程块索引，0~grid-1
//printf("blockDim.x is %d\n",blockDim.x);//线程块包含的线程数，这里就是<<<grid,block,size>>>中的block
//printf("threadIdx.x is %d\n",threadIdx.x);//每个线程块中线程的标号，0~block-1
int id = blockIdx.x*blockDim.x + threadIdx.x;//为每个线程构建唯一标号，0~grid*block-1

T temp;
T pSum = 0;
extern T __shared__ s_pi[];//数据存放在共享存储上，只有本线程块内的线程可以访问
T rnum = 1.0/num;

for(int i=id;i<num;i +=blockDim.x*gridDim.x){
	//每个线程计算的次数是总的次数（num）除以总的线程数（grid*block）
	temp = (i+0.5f)*rnum;
	pSum += 4.0f/(1+temp*temp);
}

s_pi[threadIdx.x] = pSum*rnum;//每个线程块中的线程会把自己计算得到的s_pi独立存储在本块的共享存储上
__syncthreads();//等待本块所有线程计算完毕

for(int i = (blockDim.x>>1);i >0;i >>= 1){
//将本块内的 计算结果 进行累加
	if (threadIdx.x<i){
		s_pi[threadIdx.x] += s_pi[threadIdx.x+i];
	}
	__syncthreads();
}
//将加和的结果写到本块对应的显存中，以备reducePI2使用
if (threadIdx.x==0)
{
	d_sum[blockIdx.x]=s_pi[0];
}

//下面这段代码应该是在执行类似的算法但是结果会有很大偏差，并未找到原因^_^
//if (warpSize>63){
//	if (threadIdx.x<32){
//		s_pi[threadIdx.x] += s_pi[threadIdx.x +32];
//	}
//}
//if (threadIdx.x<16){
//	s_pi[threadIdx.x] += s_pi[threadIdx.x +16];
//printf("threadIdx.x 16 is %d\n",threadIdx.x);
//}
//if (threadIdx.x<8){
//	s_pi[threadIdx.x] += s_pi[threadIdx.x +8];
//printf("threadIdx.x 8 is %d\n",threadIdx.x);
//}
//if (threadIdx.x<4){
//	s_pi[threadIdx.x] += s_pi[threadIdx.x +4];
//printf("threadIdx.x 4 is %d\n",threadIdx.x);
//}
//if (threadIdx.x<2){
//	s_pi[threadIdx.x] += s_pi[threadIdx.x +2];
//printf("threadIdx.x 2 is %d\n",threadIdx.x);
//}
//if (threadIdx.x<1){
//	d_sum[blockIdx.x] = s_pi[0]+s_pi[1];
//printf("threadIdx.x 1 is %d\n",threadIdx.x);
//}



}

template<typename T> __global__ void reducePI2(T* __restrict__ d_sum, int num, T* __restrict__ d_pi){
int id = threadIdx.x;//这个函数的线程块只有一个，线程数是grid，这里依然用id作为索引名
extern T __shared__ s_sum[];//这个是共享内存中的，只有块内可见
s_sum[id]=d_sum[id];//把显存中的数据装载进来
__syncthreads();//等待装载完成

for(int i = (blockDim.x>>1);i>0;i >>=1)
//仍然采用半对半折和的方法对本块内所有线程中的s_sum进行求和
{
	if (id<i){
		s_sum[id] += s_sum[id+i];	
	}
	__syncthreads();//等待求和完成
}
//将求和结果写入显存，使得cpu主机端可见
if(threadIdx.x==0)
{
	*d_pi =s_sum[0];
}
//if (warpSize>63){
//	if (threadIdx.x<32){
//		s_sum[threadIdx.x] += s_sum[threadIdx.x +32];
//	}
//}
//if (threadIdx.x<16){
//	s_sum[threadIdx.x] += s_sum[threadIdx.x +16];
//}//
//if (threadIdx.x<8){
//	s_sum[threadIdx.x] += s_sum[threadIdx.x +8];
//}
//if (threadIdx.x<4){
//	s_sum[threadIdx.x] += s_sum[threadIdx.x +4];
//}
//if (threadIdx.x<2){
//	s_sum[threadIdx.x] += s_sum[threadIdx.x +2];
//}
//if (threadIdx.x<1){
//	*d_pi = s_sum[0]+s_sum[1];
//}

}

//**********************************************//
//以下代码在主机上编译

template <typename T> T reducePI(int num){

int grid = 1024;//用来调整线程块的数量

T *tmp;
hipMalloc((void**)&tmp,grid*sizeof(T));//在设备存储器（显存）上开辟grid*sizeof(T)大小的空间，主机上的指针tmp指向该空间
reducePI1<<<grid,256,256*sizeof(T)>>>(tmp,num);//调用reducePI1
//参数表示有grid个线程块，每个线程块有256个线程，每个线程块使用256*size大小的共享存储器（只有块内可以访问）

//执行之后，会在tmp为首的显存中存储grid 个中间结果
//printf("%d\n",__LINE__);//显示代码所在行号，不知会有什么用
T *d_PI;
hipMalloc((void**)&d_PI,sizeof(T));//显存中为π的计算结果开辟空间

reducePI2<<<1,grid,grid*sizeof(T)>>>(tmp,grid,d_PI);//只有一个线程块，有grid个线程
//执行后在显存中d_PI的位置存放最后结果
T pi;//这是在主机内存上的空间
hipMemcpy(&pi,d_PI,sizeof(T),hipMemcpyDeviceToHost);//从显存中将数据拷贝出来
hipFree(tmp);//释放相应的显存空间
hipFree(d_PI);

return pi;
}

template <typename T> T cpuPI(int num){

T sum = 0.0f;
T temp;
for (int i=0;i<num;i++)
{
	temp =(i+0.5f)/num;
	sum += 4/(1+temp*temp);
}
return sum/num;

}


int main(){
printf("test for compell \n");
clock_t start, finish;//用来计时
float costtime;
start = clock(); 
//************
printf("cpu pi is  %f\n",cpuPI<float>(1000000));//调用普通的串行循环计算 π
//*************
finish = clock();
costtime = (float)(finish - start) / CLOCKS_PER_SEC; //单位是秒
printf("costtime of CPU is %f\n",costtime);

start = clock();
//************
printf("gpu pi is %f\n",reducePI<float>(1000000));//调用主机上的并行计算函数
//************
finish = clock();
costtime = (float)(finish - start) / CLOCKS_PER_SEC; 
printf("costtime of GPU is %f\n",costtime);
return 0;
}
