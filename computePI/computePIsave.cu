
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <time.h>
//#include <cutil_inline.h>
using namespace std;

//*****************************************//
//���������ֽ����豸�ϱ��� ��__global__��ʶ��
template<typename T> __global__ void reducePI1(T* __restrict__ d_sum, int num){
//__restrict__ ��˵��ֻ�������ж�ȡ�����ݣ�����ʲô�����أ�
//printf("blockIdx.x is %d\n",blockIdx.x);//�߳̿�������0~grid-1
//printf("blockDim.x is %d\n",blockDim.x);//�߳̿�������߳������������<<<grid,block,size>>>�е�block
//printf("threadIdx.x is %d\n",threadIdx.x);//ÿ���߳̿����̵߳ı�ţ�0~block-1
int id = blockIdx.x*blockDim.x + threadIdx.x;//Ϊÿ���̹߳���Ψһ��ţ�0~grid*block-1

T temp;
T pSum = 0;
extern T __shared__ s_pi[];//���ݴ���ڹ���洢�ϣ�ֻ�б��߳̿��ڵ��߳̿��Է���
T rnum = 1.0/num;

for(int i=id;i<num;i +=blockDim.x*gridDim.x){
	//ÿ���̼߳���Ĵ������ܵĴ�����num�������ܵ��߳�����grid*block��
	temp = (i+0.5f)*rnum;
	pSum += 4.0f/(1+temp*temp);
}

s_pi[threadIdx.x] = pSum*rnum;//ÿ���߳̿��е��̻߳���Լ�����õ���s_pi�����洢�ڱ���Ĺ���洢��
__syncthreads();//�ȴ����������̼߳������

for(int i = (blockDim.x>>1);i >0;i >>= 1){
//�������ڵ� ������ �����ۼ�
	if (threadIdx.x<i){
		s_pi[threadIdx.x] += s_pi[threadIdx.x+i];
	}
	__syncthreads();
}
//���Ӻ͵Ľ��д�������Ӧ���Դ��У��Ա�reducePI2ʹ��
if (threadIdx.x==0)
{
	d_sum[blockIdx.x]=s_pi[0];
}

//������δ���Ӧ������ִ�����Ƶ��㷨���ǽ�����кܴ�ƫ���δ�ҵ�ԭ��^_^
//if (warpSize>63){
//	if (threadIdx.x<32){
//		s_pi[threadIdx.x] += s_pi[threadIdx.x +32];
//	}
//}
//if (threadIdx.x<16){
//	s_pi[threadIdx.x] += s_pi[threadIdx.x +16];
//printf("threadIdx.x 16 is %d\n",threadIdx.x);
//}
//if (threadIdx.x<8){
//	s_pi[threadIdx.x] += s_pi[threadIdx.x +8];
//printf("threadIdx.x 8 is %d\n",threadIdx.x);
//}
//if (threadIdx.x<4){
//	s_pi[threadIdx.x] += s_pi[threadIdx.x +4];
//printf("threadIdx.x 4 is %d\n",threadIdx.x);
//}
//if (threadIdx.x<2){
//	s_pi[threadIdx.x] += s_pi[threadIdx.x +2];
//printf("threadIdx.x 2 is %d\n",threadIdx.x);
//}
//if (threadIdx.x<1){
//	d_sum[blockIdx.x] = s_pi[0]+s_pi[1];
//printf("threadIdx.x 1 is %d\n",threadIdx.x);
//}



}

template<typename T> __global__ void reducePI2(T* __restrict__ d_sum, int num, T* __restrict__ d_pi){
int id = threadIdx.x;//����������߳̿�ֻ��һ�����߳�����grid��������Ȼ��id��Ϊ������
extern T __shared__ s_sum[];//����ǹ����ڴ��еģ�ֻ�п��ڿɼ�
s_sum[id]=d_sum[id];//���Դ��е�����װ�ؽ���
__syncthreads();//�ȴ�װ�����

for(int i = (blockDim.x>>1);i>0;i >>=1)
//��Ȼ���ð�԰��ۺ͵ķ����Ա����������߳��е�s_sum�������
{
	if (id<i){
		s_sum[id] += s_sum[id+i];	
	}
	__syncthreads();//�ȴ�������
}
//����ͽ��д���Դ棬ʹ��cpu�����˿ɼ�
if(threadIdx.x==0)
{
	*d_pi =s_sum[0];
}
//if (warpSize>63){
//	if (threadIdx.x<32){
//		s_sum[threadIdx.x] += s_sum[threadIdx.x +32];
//	}
//}
//if (threadIdx.x<16){
//	s_sum[threadIdx.x] += s_sum[threadIdx.x +16];
//}//
//if (threadIdx.x<8){
//	s_sum[threadIdx.x] += s_sum[threadIdx.x +8];
//}
//if (threadIdx.x<4){
//	s_sum[threadIdx.x] += s_sum[threadIdx.x +4];
//}
//if (threadIdx.x<2){
//	s_sum[threadIdx.x] += s_sum[threadIdx.x +2];
//}
//if (threadIdx.x<1){
//	*d_pi = s_sum[0]+s_sum[1];
//}

}

//**********************************************//
//���´����������ϱ���

template <typename T> T reducePI(int num){

int grid = 1024;//���������߳̿������

T *tmp;
hipMalloc((void**)&tmp,grid*sizeof(T));//���豸�洢�����Դ棩�Ͽ���grid*sizeof(T)��С�Ŀռ䣬�����ϵ�ָ��tmpָ��ÿռ�
reducePI1<<<grid,256,256*sizeof(T)>>>(tmp,num);//����reducePI1
//������ʾ��grid���߳̿飬ÿ���߳̿���256���̣߳�ÿ���߳̿�ʹ��256*size��С�Ĺ���洢����ֻ�п��ڿ��Է��ʣ�

//ִ��֮�󣬻���tmpΪ�׵��Դ��д洢grid ���м���
//printf("%d\n",__LINE__);//��ʾ���������кţ���֪����ʲô��
T *d_PI;
hipMalloc((void**)&d_PI,sizeof(T));//�Դ���Ϊ�еļ��������ٿռ�

reducePI2<<<1,grid,grid*sizeof(T)>>>(tmp,grid,d_PI);//ֻ��һ���߳̿飬��grid���߳�
//ִ�к����Դ���d_PI��λ�ô�������
T pi;//�����������ڴ��ϵĿռ�
hipMemcpy(&pi,d_PI,sizeof(T),hipMemcpyDeviceToHost);//���Դ��н����ݿ�������
hipFree(tmp);//�ͷ���Ӧ���Դ�ռ�
hipFree(d_PI);

return pi;
}

template <typename T> T cpuPI(int num){

T sum = 0.0f;
T temp;
for (int i=0;i<num;i++)
{
	temp =(i+0.5f)/num;
	sum += 4/(1+temp*temp);
}
return sum/num;

}


int main(){
printf("test for compell \n");
clock_t start, finish;//������ʱ
float costtime;
start = clock(); 
//************
printf("cpu pi is  %f\n",cpuPI<float>(1000000));//������ͨ�Ĵ���ѭ������ ��
//*************
finish = clock();
costtime = (float)(finish - start) / CLOCKS_PER_SEC; //��λ����
printf("costtime of CPU is %f\n",costtime);

start = clock();
//************
printf("gpu pi is %f\n",reducePI<float>(1000000));//���������ϵĲ��м��㺯��
//************
finish = clock();
costtime = (float)(finish - start) / CLOCKS_PER_SEC; 
printf("costtime of GPU is %f\n",costtime);
return 0;
}
