
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <time.h>
//#include <cutil_inline.h>
using namespace std;

template<typename T> __global__ void reducePI1(T* __restrict__ d_sum, int num){
//printf("blockIdx.x is %d\n",blockIdx.x);
//printf("blockDim.x is %d\n",blockDim.x);
//printf("threadIdx.x is %d\n",threadIdx.x);
int id = blockIdx.x*blockDim.x + threadIdx.x;

T temp;
T pSum = 0;
extern T __shared__ s_pi[];
T rnum = 1.0/num;

for(int i=id;i<num;i +=blockDim.x*gridDim.x){
temp = (i+0.5f)*rnum;
pSum += 4.0f/(1+temp*temp);
}

s_pi[threadIdx.x] = pSum*rnum;
__syncthreads();

for(int i = (blockDim.x>>1);i >0;i >>= 1){
//for(int i = (blockDim.x>>1);i >= 0;i >>=1){
	if (threadIdx.x<i){
		s_pi[threadIdx.x] += s_pi[threadIdx.x+i];
	}
	__syncthreads();
}

if (threadIdx.x==0)
{
	d_sum[blockIdx.x]=s_pi[0];
}

//if (warpSize>63){
//	if (threadIdx.x<32){
//		s_pi[threadIdx.x] += s_pi[threadIdx.x +32];
//	}
//}
//if (threadIdx.x<16){
//	s_pi[threadIdx.x] += s_pi[threadIdx.x +16];
//printf("threadIdx.x 16 is %d\n",threadIdx.x);
//}
//if (threadIdx.x<8){
//	s_pi[threadIdx.x] += s_pi[threadIdx.x +8];
//printf("threadIdx.x 8 is %d\n",threadIdx.x);
//}
//if (threadIdx.x<4){
//	s_pi[threadIdx.x] += s_pi[threadIdx.x +4];
//printf("threadIdx.x 4 is %d\n",threadIdx.x);
//}
//if (threadIdx.x<2){
//	s_pi[threadIdx.x] += s_pi[threadIdx.x +2];
//printf("threadIdx.x 2 is %d\n",threadIdx.x);
//}
//if (threadIdx.x<1){
//	d_sum[blockIdx.x] = s_pi[0]+s_pi[1];
//printf("threadIdx.x 1 is %d\n",threadIdx.x);
//}



}

template<typename T> __global__ void reducePI2(T* __restrict__ d_sum, int num, T* __restrict__ d_pi){
int id = threadIdx.x;
extern T __shared__ s_sum[];
s_sum[id]=d_sum[id];
__syncthreads();

for(int i = (blockDim.x>>1);i>0;i >>=1)
//for(int i = (blockDim.x>>1);i>=0;i >>=1)
{
	if (id<i){
		s_sum[id] += s_sum[id+i];	
	}
	__syncthreads();
}

if(threadIdx.x==0)
{
	*d_pi =s_sum[0];
}
//if (warpSize>63){
//	if (threadIdx.x<32){
//		s_sum[threadIdx.x] += s_sum[threadIdx.x +32];
//	}
//}
//if (threadIdx.x<16){
//	s_sum[threadIdx.x] += s_sum[threadIdx.x +16];
//}//
//if (threadIdx.x<8){
//	s_sum[threadIdx.x] += s_sum[threadIdx.x +8];
//}
//if (threadIdx.x<4){
//	s_sum[threadIdx.x] += s_sum[threadIdx.x +4];
//}
//if (threadIdx.x<2){
//	s_sum[threadIdx.x] += s_sum[threadIdx.x +2];
//}
//if (threadIdx.x<1){
//	*d_pi = s_sum[0]+s_sum[1];
//}

}

template <typename T> T reducePI(int num){

int grid = 1024;

T *tmp;
hipMalloc((void**)&tmp,grid*sizeof(T));
reducePI1<<<grid,256,256*sizeof(T)>>>(tmp,num);


//printf("%d\n",__LINE__);
T *d_PI;
hipMalloc((void**)&d_PI,sizeof(T));

reducePI2<<<1,grid,grid*sizeof(T)>>>(tmp,grid,d_PI);
T pi;
hipMemcpy(&pi,d_PI,sizeof(T),hipMemcpyDeviceToHost);
hipFree(tmp);
hipFree(d_PI);

return pi;
}

template <typename T> T cpuPI(int num){

T sum = 0.0f;
T temp;
for (int i=0;i<num;i++)
{
	temp =(i+0.5f)/num;
	sum += 4/(1+temp*temp);
}
return sum/num;

}


int main(){
printf("test for compell \n");
clock_t start, finish;
float costtime;
start = clock(); 
//************
printf("cpu pi is  %f\n",cpuPI<float>(1000000));
//*************
finish = clock();
costtime = (float)(finish - start) / CLOCKS_PER_SEC; 
printf("costtime of CPU is %f\n",costtime);

start = clock();
//************
printf("gpu pi is %f\n",reducePI<float>(1000000));
//************
finish = clock();
costtime = (float)(finish - start) / CLOCKS_PER_SEC; 
printf("costtime of GPU is %f\n",costtime);
return 0;
}
